#include "hip/hip_runtime.h"
//==================================================
// bvh.cu
//
//  Copyright (c) 2016 Benjamin Hepp.
//  Author: Benjamin Hepp
//  Created on: Jan 16, 2017
//==================================================
#include <ait/cuda_utils.h>
#include <vector>
#include <cstdio>
#include <deque>
#include <iostream>
#include <ait/utilities.h>
#include "bvh.cuh"

using std::printf;

namespace bvh {

template <typename FloatT>
CudaTree<FloatT>* CudaTree<FloatT>::createCopyFromHostTree(
    NodeType* root,
    const std::size_t num_of_nodes,
    const std::size_t tree_depth) {
  CudaTree* cuda_tree = new CudaTree(tree_depth);
  const std::size_t memory_size = sizeof(NodeType) * num_of_nodes;
  std::cout << "Allocating " << (memory_size / 1024. / 1024.) << " MB of GPU memory" << std::endl;
  cuda_tree->d_nodes_ = ait::CudaUtils::template allocate<NodeType>(num_of_nodes);
  std::deque<NodeType*> node_queue;
  node_queue.push_front(root);
  std::size_t node_counter = 0;
  std::size_t copied_node_counter = 0;
  const std::size_t report_threshold = num_of_nodes / 20;
  std::size_t report_counter = 0;
  const std::size_t node_cache_size = num_of_nodes / 20;
  std::vector<CudaNode<FloatT>> node_cache;
  node_cache.reserve(node_cache_size);
  while (!node_queue.empty()) {
    NodeType* node = node_queue.back();
    node_queue.pop_back();

    CudaNode<FloatT> cuda_node;
    cuda_node.bounding_box_ = node->bounding_box_;
    cuda_node.ptr_ = static_cast<void*>(node);
    AIT_ASSERT(cuda_node.ptr_ != nullptr);
    if (node->hasLeftChild()) {
      node_queue.push_front(node->getLeftChild());
      const std::size_t left_child_index = node_counter + node_queue.size();
      cuda_node.left_child_ = &cuda_tree->d_nodes_[left_child_index];
    }
    else {
      cuda_node.left_child_ = nullptr;
    }
    if (node->hasRightChild()) {
      node_queue.push_front(node->getRightChild());
      std::size_t right_child_index = node_counter + node_queue.size();
      cuda_node.right_child_ = &cuda_tree->d_nodes_[right_child_index];
    }
    else {
      cuda_node.right_child_ = nullptr;
    }
//    ait::CudaUtils::copyToDevice(cuda_node, &tree->d_nodes_[node_counter]);
    node_cache.push_back(cuda_node);
    if (node_cache.size() == node_cache_size) {
      ait::CudaUtils::copyArrayToDevice(node_cache, &cuda_tree->d_nodes_[copied_node_counter]);
      copied_node_counter += node_cache.size();
      node_cache.clear();
    }
    ++node_counter;
    ++report_counter;
    if (report_counter >= report_threshold) {
      std::cout << "Copied " << node_counter << " nodes [" << (100 * node_counter / (FloatT)num_of_nodes) << " %]" << std::endl;
      report_counter = 0;
    }
  }
  return cuda_tree;
}

template <typename FloatT>
__device__ bool intersectsIterativeCuda(
    const typename CudaTree<FloatT>::CudaIntersectionData data,
    typename CudaTree<FloatT>::CudaIntersectionIterativeStackEntry* stack,
    std::size_t stack_size,
    typename CudaTree<FloatT>::CudaIntersectionResult* d_result) {
  while (stack_size > 0) {
    typename CudaTree<FloatT>::CudaIntersectionIterativeStackEntry& entry = stack[stack_size - 1];
    if (entry.state == CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::NotVisited) {
      entry.intersects = false;
      bool outside_bounding_box = entry.node->getBoundingBox().isOutside(data.ray.origin);
      CudaVector3<FloatT> intersection;
      FloatT intersection_dist_sq;
      bool early_break = false;
      if (outside_bounding_box) {
        // Check if ray intersects current node
        const bool intersects = entry.node->getBoundingBox().intersectsCuda(data.ray, &intersection);
    //      std::cout << "intersects: " << intersects << std::endl;
        if (intersects) {
          intersection_dist_sq = (data.ray.origin - intersection).squaredNorm();
          if (intersection_dist_sq > d_result->dist_sq) {
            early_break = true;
          }
        }
        else {
          early_break = true;
        }
      }
      if (early_break) {
        --stack_size;
      }
      else {
        if (entry.node->isLeaf()) {
          if (!outside_bounding_box) {
            // If already inside the bounding box we want the intersection point to be the start of the ray.
            intersection = data.ray.origin;
            intersection_dist_sq = 0;
          }
          d_result->intersection = intersection;
          d_result->node = static_cast<void*>(entry.node->getPtr());
          d_result->depth = entry.depth;
          d_result->dist_sq = intersection_dist_sq;
          entry.intersects = true;
          --stack_size;
        }
        else {
          if (entry.node->hasLeftChild()) {
            stack[stack_size].node = entry.node->getLeftChild();
            stack[stack_size].depth = entry.depth + 1;
            stack[stack_size].state = CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::NotVisited;
            ++stack_size;
            entry.state = CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::PushedLeftChild;
          }
        }
      }
    }
    else if (entry.state == CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::PushedLeftChild) {
      // This stack entry was processed before. The next stack entries contain the child results
      entry.intersects = stack[stack_size + 0].intersects;
      if (entry.node->hasRightChild()) {
        stack[stack_size].node = entry.node->getRightChild();
        stack[stack_size].depth = entry.depth + 1;
        stack[stack_size].state = CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::NotVisited;
        ++stack_size;
        entry.state = CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::PushedRightChild;
      }
    }
    else if (entry.state == CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::PushedRightChild) {
      // This stack entry was processed before. The next stack entries contain the child results
      if (!entry.intersects) {
        entry.intersects = stack[stack_size].intersects;
      }
      --stack_size;
    }
  }
  return stack[0].intersects;
}

#if WITH_CUDA_RECURSION
template <typename FloatT>
__device__ bool intersectsRecursiveCuda(
    const typename CudaTree<FloatT>::CudaIntersectionData data,
    typename CudaTree<FloatT>::NodeType* cur_node,
    const std::size_t cur_depth,
    typename CudaTree<FloatT>::CudaIntersectionResult* d_result) {
  bool outside_bounding_box = cur_node->getBoundingBox().isOutside(data.ray.origin);
  CudaVector3<FloatT> intersection;
  FloatT intersection_dist_sq;
  if (outside_bounding_box) {
    // Check if ray intersects current node
    const bool intersects = cur_node->getBoundingBox().intersectsCuda(data.ray, &intersection);
    if (intersects) {
      intersection_dist_sq = (data.ray.origin - intersection).squaredNorm();
      if (intersection_dist_sq > d_result->dist_sq) {
        return false;
      }
    }
    else {
      return false;
    }
  }
  if (cur_node->isLeaf()) {
    if (!outside_bounding_box) {
      // If already inside the bounding box we want the intersection point to be the start of the ray.
      intersection = data.ray.origin;
      intersection_dist_sq = 0;
    }
    d_result->intersection = intersection;
    d_result->node = static_cast<void*>(cur_node->getPtr());
    d_result->depth = cur_depth;
    d_result->dist_sq = intersection_dist_sq;
    return true;
  }

  bool intersects_left = false;
  bool intersects_right = false;
  if (cur_node->hasLeftChild()) {
    intersects_left = intersectsRecursiveCuda<FloatT>(data, cur_node->getLeftChild(), cur_depth + 1, d_result);
  }
  if (cur_node->hasRightChild()) {
    intersects_right = intersectsRecursiveCuda<FloatT>(data, cur_node->getRightChild(), cur_depth + 1, d_result);
  }
  return intersects_left || intersects_right;
}
#endif

#if WITH_CUDA_RECURSION
template <typename FloatT>
__global__ void intersectsRecursiveCudaKernel(
    const typename CudaTree<FloatT>::CudaRayType* d_rays,
    const std::size_t num_of_rays,
    const FloatT min_range,
    const FloatT max_range,
    typename CudaTree<FloatT>::NodeType* d_root,
    typename CudaTree<FloatT>::CudaIntersectionResult* d_results) {
  const std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_of_rays) {
    const typename CudaTree<FloatT>::CudaRayType& ray = d_rays[index];
    typename CudaTree<FloatT>::CudaIntersectionData data;
    data.ray.origin = ray.origin;
    data.ray.direction = ray.direction;
    data.ray.inv_direction = ray.direction.cwiseInverse();
    data.min_range_sq = min_range * min_range;
    typename CudaTree<FloatT>::CudaIntersectionResult& result = d_results[index];
    result.dist_sq = max_range > 0 ? max_range * max_range : FLT_MAX;
    result.node = nullptr;
    const std::size_t cur_depth = 0;
  //  printf("Calling intersectsRecursiveCuda\n");
    intersectsRecursiveCuda<FloatT>(data, d_root, cur_depth, &result);
  }
}
#endif

template <typename FloatT>
__global__ void intersectsIterativeCudaKernel(
    const typename CudaTree<FloatT>::CudaRayType* d_rays,
    const std::size_t num_of_rays,
    const FloatT min_range,
    const FloatT max_range,
    typename CudaTree<FloatT>::NodeType* d_root,
    typename CudaTree<FloatT>::CudaIntersectionIterativeStackEntry* stacks,
    const std::size_t max_stack_size,
    typename CudaTree<FloatT>::CudaIntersectionResult* d_results) {
  const std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_of_rays) {
    const typename CudaTree<FloatT>::CudaRayType& ray = d_rays[index];
    typename CudaTree<FloatT>::CudaIntersectionData data;
    data.ray.origin = ray.origin;
    data.ray.direction = ray.direction;
    data.ray.inv_direction = ray.direction.cwiseInverse();
    data.min_range_sq = min_range * min_range;
    typename CudaTree<FloatT>::CudaIntersectionResult& result = d_results[index];
    result.dist_sq = max_range > 0 ? max_range * max_range : FLT_MAX;
    result.node = nullptr;
    typename CudaTree<FloatT>::CudaIntersectionIterativeStackEntry* stack = &stacks[index * max_stack_size];
    stack[0].depth = 0;
    stack[0].node = d_root;
    const std::size_t stack_size = 1;
    for (std::size_t i = 0; i < max_stack_size; ++i) {
      stack[i].state = CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::NotVisited;
    }
    intersectsIterativeCuda<FloatT>(data, stack, stack_size, &result);
  }
}

template <typename FloatT>
__device__ CudaVector3<FloatT> getCameraRay(
    const CudaMatrix4x4<FloatT>& intrinsics,
    const FloatT x,
    const FloatT y) {
  CudaVector3<FloatT> ray_direction;
  ray_direction(0) = (x - intrinsics(0, 2)) / intrinsics(0, 0);
  ray_direction(1) = (y - intrinsics(1, 2)) / intrinsics(1, 1);
  ray_direction(2) = 1;
  return ray_direction;
}

template <typename FloatT>
__device__ CudaRay<FloatT> getCameraRay(
    const CudaMatrix4x4<FloatT>& intrinsics,
    const CudaMatrix3x4<FloatT>& extrinsics,
    const FloatT x,
    const FloatT y) {
  CudaRay<FloatT> ray;
  ray.origin = extrinsics.col(3);
  CudaVector3<FloatT> direction_camera = getCameraRay(intrinsics, x, y);
  CudaMatrix3x3<FloatT> rotation = extrinsics.template block<0, 0, 3, 3>();
  ray.direction = rotation * direction_camera;
  return ray;
}

template <typename FloatT>
__global__ void raycastIterativeCudaKernel(
    const CudaMatrix4x4<FloatT> intrinsics,
    const CudaMatrix3x4<FloatT> extrinsics,
    typename CudaTree<FloatT>::CudaRayType* d_rays,
    const std::size_t num_of_rays,
    const std::size_t x_start, const std::size_t x_end,
    const std::size_t y_start, const std::size_t y_end,
    const FloatT min_range,
    const FloatT max_range,
    typename CudaTree<FloatT>::NodeType* d_root,
    typename CudaTree<FloatT>::CudaIntersectionIterativeStackEntry* stacks,
    const std::size_t max_stack_size,
    typename CudaTree<FloatT>::CudaIntersectionResult* d_results) {
  const std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const std::size_t yi = blockIdx.x;
  const std::size_t xi = threadIdx.x;
  const FloatT yf = y_start + yi;
  const FloatT xf = x_start + xi;
  if (xi < (x_end - x_start) && yi < (y_end - y_start)) {
    typename CudaTree<FloatT>::CudaRayType& ray = d_rays[index];
    ray = getCameraRay(intrinsics, extrinsics, xf, yf);
    typename CudaTree<FloatT>::CudaIntersectionData data;
    data.ray.origin = ray.origin;
    data.ray.direction = ray.direction;
    data.ray.inv_direction = ray.direction.cwiseInverse();
    data.min_range_sq = min_range * min_range;
    typename CudaTree<FloatT>::CudaIntersectionResult& result = d_results[index];
    result.dist_sq = max_range > 0 ? max_range * max_range : FLT_MAX;
    result.node = nullptr;
  //  printf("Calling intersectsRecursiveCuda\n");
    typename CudaTree<FloatT>::CudaIntersectionIterativeStackEntry* stack = &stacks[index * max_stack_size];
    stack[0].depth = 0;
    stack[0].node = d_root;
    const std::size_t stack_size = 1;
    for (std::size_t i = 0; i < max_stack_size; ++i) {
      stack[i].state = CudaTree<FloatT>::CudaIntersectionIterativeStackEntry::NotVisited;
    }
    intersectsIterativeCuda<FloatT>(data, stack, stack_size, &result);
  }
}

#if WITH_CUDA_RECURSION
template <typename FloatT>
__global__ void raycastRecursiveCudaKernel(
    const CudaMatrix4x4<FloatT> intrinsics,
    const CudaMatrix3x4<FloatT> extrinsics,
    typename CudaTree<FloatT>::CudaRayType* d_rays,
    const std::size_t num_of_rays,
    const std::size_t x_start, const std::size_t x_end,
    const std::size_t y_start, const std::size_t y_end,
    const FloatT min_range,
    const FloatT max_range,
    typename CudaTree<FloatT>::NodeType* d_root,
    typename CudaTree<FloatT>::CudaIntersectionResult* d_results) {
  const std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const std::size_t yi = blockIdx.x;
  const std::size_t xi = threadIdx.x;
  const FloatT yf = y_start + yi;
  const FloatT xf = x_start + xi;
  if (xi < (x_end - x_start) && yi < (y_end - y_start)) {
    typename CudaTree<FloatT>::CudaRayType& ray = d_rays[index];
    ray = getCameraRay(intrinsics, extrinsics, xf, yf);
    typename CudaTree<FloatT>::CudaIntersectionData data;
    data.ray.origin = ray.origin;
    data.ray.direction = ray.direction;
    data.ray.inv_direction = ray.direction.cwiseInverse();
    data.min_range_sq = min_range * min_range;
    typename CudaTree<FloatT>::CudaIntersectionResult& result = d_results[index];
    result.dist_sq = max_range > 0 ? max_range * max_range : FLT_MAX;
    result.node = nullptr;
    const std::size_t cur_depth = 0;
  //  printf("Calling intersectsRecursiveCuda\n");
    intersectsRecursiveCuda<FloatT>(data, d_root, cur_depth, &result);
  }
}

template <typename FloatT>
__global__ void raycastWithScreenCoordinatesRecursiveCudaKernel(
    const CudaMatrix4x4<FloatT> intrinsics,
    const CudaMatrix3x4<FloatT> extrinsics,
    typename CudaTree<FloatT>::CudaRayType* d_rays,
    const std::size_t num_of_rays,
    const std::size_t x_start, const std::size_t x_end,
    const std::size_t y_start, const std::size_t y_end,
    const FloatT min_range,
    const FloatT max_range,
    typename CudaTree<FloatT>::NodeType* d_root,
    typename CudaTree<FloatT>::CudaIntersectionResultWithScreenCoordinates* d_results_with_screen_coordinates) {
  const std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const std::size_t yi = blockIdx.x;
  const std::size_t xi = threadIdx.x;
  const FloatT yf = y_start + yi;
  const FloatT xf = x_start + xi;
  if (xi < (x_end - x_start) && yi < (y_end - y_start)) {
    typename CudaTree<FloatT>::CudaRayType& ray = d_rays[index];
    ray = getCameraRay(intrinsics, extrinsics, xf, yf);
    typename CudaTree<FloatT>::CudaIntersectionData data;
    data.ray.origin = ray.origin;
    data.ray.direction = ray.direction;
    data.ray.inv_direction = ray.direction.cwiseInverse();
    data.min_range_sq = min_range * min_range;
    typename CudaTree<FloatT>::CudaIntersectionResultWithScreenCoordinates& result = d_results_with_screen_coordinates[index];
    result.intersection_result.dist_sq = max_range > 0 ? max_range * max_range : FLT_MAX;
    result.intersection_result.node = nullptr;
    const std::size_t cur_depth = 0;
  //  printf("Calling intersectsRecursiveCuda\n");
    result.screen_coordinates(0) = xf;
    result.screen_coordinates(1) = yf;
    intersectsRecursiveCuda<FloatT>(data, d_root, cur_depth, &result.intersection_result);
  }
}
#endif

#if WITH_CUDA_RECURSION
template <typename FloatT>
std::vector<typename CudaTree<FloatT>::CudaIntersectionResult>
CudaTree<FloatT>::intersectsRecursive(const std::vector<CudaRayType>& rays, FloatT min_range /*= 0*/, FloatT max_range /*= -1*/) {
  reserveDeviceRaysAndResults(rays.size());
  ait::CudaUtils::copyArrayToDevice(rays, d_rays_);
  const std::size_t grid_size = (rays.size() + kThreadsPerBlock - 1) / kThreadsPerBlock;
  const std::size_t block_size = std::min(kThreadsPerBlock, rays.size());
  AIT_ASSERT(grid_size > 0);
  AIT_ASSERT(block_size > 0);
//  std::cout << "grid_size=" << grid_size << ", block_size=" << block_size << std::endl;
  intersectsRecursiveCudaKernel<FloatT><<<grid_size, block_size>>>(
      d_rays_, rays.size(),
      min_range, max_range,
      getRoot(),
      d_results_);
  CUDA_DEVICE_SYNCHRONIZE();
  CUDA_CHECK_ERROR();
  std::vector<CudaIntersectionResult> cuda_results(rays.size());
  ait::CudaUtils::copyArrayFromDevice(d_results_, &cuda_results);
  return cuda_results;
}
#endif

template <typename FloatT>
std::vector<typename CudaTree<FloatT>::CudaIntersectionResult>
CudaTree<FloatT>::intersectsIterative(const std::vector<CudaRayType>& rays, FloatT min_range /*= 0*/, FloatT max_range /*= -1*/) {
  reserveDeviceRaysAndResults(rays.size());
  ait::CudaUtils::copyArrayToDevice(rays, d_rays_);
  const std::size_t grid_size = (rays.size() + kThreadsPerBlock - 1) / kThreadsPerBlock;
  const std::size_t block_size = std::min(kThreadsPerBlock, rays.size());
  AIT_ASSERT(grid_size > 0);
  AIT_ASSERT(block_size > 0);
//  std::cout << "grid_size=" << grid_size << ", block_size=" << block_size << std::endl;
  intersectsIterativeCudaKernel<FloatT><<<grid_size, block_size>>>(
      d_rays_, rays.size(),
      min_range, max_range,
      getRoot(),
      d_stacks_,
      tree_depth_ + 1,
      d_results_);
  CUDA_DEVICE_SYNCHRONIZE();
  CUDA_CHECK_ERROR();
  std::vector<CudaIntersectionResult> cuda_results(rays.size());
  ait::CudaUtils::copyArrayFromDevice(d_results_, &cuda_results);
  return cuda_results;
}

#if WITH_CUDA_RECURSION
template <typename FloatT>
std::vector<typename CudaTree<FloatT>::CudaIntersectionResult>
CudaTree<FloatT>::raycastRecursive(
    const CudaMatrix4x4<FloatT>& intrinsics,
    const CudaMatrix3x4<FloatT>& extrinsics,
    const std::size_t x_start, const std::size_t x_end,
    const std::size_t y_start, const std::size_t y_end,
    FloatT min_range /*= 0*/, FloatT max_range /*= -1*/,
    const bool fail_on_error /*= false*/) {
  const std::size_t num_of_rays = (y_end - y_start) * (x_end - x_start);
  reserveDeviceRaysAndResults(num_of_rays);
  const std::size_t grid_size = y_end - y_start;
  const std::size_t block_size = x_end - x_start;
  AIT_ASSERT(grid_size > 0);
  AIT_ASSERT(block_size > 0);
  //  std::cout << "grid_size=" << grid_size << ", block_size=" << block_size << std::endl;
  NodeType* root = getRoot();
  raycastRecursiveCudaKernel<FloatT><<<grid_size, block_size>>>(
      intrinsics, extrinsics,
      d_rays_, num_of_rays,
      x_start, x_end,
      y_start, y_end,
      min_range, max_range,
      root,
      d_results_);
  if (fail_on_error) {
    CUDA_DEVICE_SYNCHRONIZE();
    CUDA_CHECK_ERROR();
  }
  else {
    hipError_t err = hipDeviceSynchronize();
    if (hipSuccess != err) { \
      fprintf(stderr, "CUDA error in file '%s' in line %i: %s\n",
          __FILE__, __LINE__, hipGetErrorString(err));
      throw ait::CudaError(err);
    }
  }
  std::vector<CudaIntersectionResult> cuda_results(num_of_rays);
  ait::CudaUtils::copyArrayFromDevice(d_results_, &cuda_results);
  return cuda_results;
}

template <typename FloatT>
std::vector<typename CudaTree<FloatT>::CudaIntersectionResultWithScreenCoordinates>
CudaTree<FloatT>::raycastWithScreenCoordinatesRecursive(
    const CudaMatrix4x4<FloatT>& intrinsics,
    const CudaMatrix3x4<FloatT>& extrinsics,
    const std::size_t x_start, const std::size_t x_end,
    const std::size_t y_start, const std::size_t y_end,
    FloatT min_range /*= 0*/, FloatT max_range /*= -1*/,
    const bool fail_on_error /*= false*/) {
  const std::size_t num_of_rays = (y_end - y_start) * (x_end - x_start);
  reserveDeviceRaysAndResults(num_of_rays);
  const std::size_t grid_size = y_end - y_start;
  const std::size_t block_size = x_end - x_start;
  AIT_ASSERT(grid_size > 0);
  AIT_ASSERT(block_size > 0);
  //  std::cout << "grid_size=" << grid_size << ", block_size=" << block_size << std::endl;
  NodeType* root = getRoot();
  raycastWithScreenCoordinatesRecursiveCudaKernel<FloatT><<<grid_size, block_size>>>(
      intrinsics, extrinsics,
      d_rays_, num_of_rays,
      x_start, x_end,
      y_start, y_end,
      min_range, max_range,
      root,
      d_results_with_screen_coordinates_);
  if (fail_on_error) {
    CUDA_DEVICE_SYNCHRONIZE();
    CUDA_CHECK_ERROR();
  }
  else {
    hipError_t err = hipDeviceSynchronize();
    if (hipSuccess != err) { \
      fprintf(stderr, "CUDA error in file '%s' in line %i: %s\n",
          __FILE__, __LINE__, hipGetErrorString(err));
      throw ait::CudaError(err);
    }
  }
  std::vector<CudaIntersectionResultWithScreenCoordinates> cuda_results(num_of_rays);
  ait::CudaUtils::copyArrayFromDevice(d_results_with_screen_coordinates_, &cuda_results);
  return cuda_results;
}
#endif

template <typename FloatT>
std::vector<typename CudaTree<FloatT>::CudaIntersectionResult>
CudaTree<FloatT>::raycastIterative(
    const CudaMatrix4x4<FloatT>& intrinsics,
    const CudaMatrix3x4<FloatT>& extrinsics,
    const std::size_t x_start, const std::size_t x_end,
    const std::size_t y_start, const std::size_t y_end,
    FloatT min_range /*= 0*/, FloatT max_range /*= -1*/) {
  const std::size_t num_of_rays = (y_end - y_start) * (x_end - x_start);
  reserveDeviceRaysAndResults(num_of_rays);
  const std::size_t grid_size = y_end - y_start;
  const std::size_t block_size = x_end - x_start;
  AIT_ASSERT(grid_size > 0);
  AIT_ASSERT(block_size > 0);
  //  std::cout << "grid_size=" << grid_size << ", block_size=" << block_size << std::endl;
  NodeType* root = getRoot();
  raycastIterativeCudaKernel<FloatT><<<grid_size, block_size>>>(
      intrinsics,
      extrinsics,
      d_rays_, num_of_rays,
      x_start, x_end,
      y_start, y_end,
      min_range, max_range,
      root,
      d_stacks_,
      tree_depth_ + 1,
      d_results_);
  CUDA_DEVICE_SYNCHRONIZE();
  CUDA_CHECK_ERROR();
  std::vector<CudaIntersectionResult> cuda_results(num_of_rays);
  ait::CudaUtils::copyArrayFromDevice(d_results_, &cuda_results);
  return cuda_results;
}

template <typename FloatT>
void CudaTree<FloatT>::reserveDeviceRaysAndResults(const std::size_t num_of_rays) {
  if (num_of_rays > d_rays_size_) {
    if (d_rays_ != nullptr) {
      ait::CudaUtils::deallocate(&d_rays_);
    }
    d_rays_ = ait::CudaUtils::template allocate<CudaRayType>(num_of_rays);
    d_rays_size_ = num_of_rays;
  }
  if (num_of_rays > d_results_size_) {
    if (d_results_ != nullptr) {
      ait::CudaUtils::deallocate(&d_results_);
    }
    d_results_ = ait::CudaUtils::template allocate<CudaIntersectionResult>(num_of_rays);
    d_results_with_screen_coordinates_ = ait::CudaUtils::template allocate<CudaIntersectionResultWithScreenCoordinates>(num_of_rays);
    d_results_size_ = num_of_rays;
  }
  if (num_of_rays > d_stacks_size_) {
    if (d_stacks_ != nullptr) {
      ait::CudaUtils::deallocate(&d_stacks_);
    }
    d_stacks_ = ait::CudaUtils::template allocate<CudaIntersectionIterativeStackEntry>(num_of_rays * (tree_depth_ + 1));
    d_stacks_size_ = num_of_rays;
  }
}

template
class CudaTree<float>;

}
